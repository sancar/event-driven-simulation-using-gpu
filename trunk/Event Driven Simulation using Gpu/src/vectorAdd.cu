#include "hip/hip_runtime.h"
﻿//============================================================================
// Name        : Event.cpp
// Author      : Ervin Domazet - M.Sancar Koyunlu
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================
#define FEL_SIZE 10//can be made larger
#include <iostream>
#include <fstream>
#include <string.h>
#include <stdlib.h>
#include "InputVector.h"

#ifndef GATES
#define GATES
#include "Gates/BaseGate.h"
#endif

#include "Input/MapReader.h"
#include "FutureEventList.h"
#include "FutureEvent.h"
#include <vector>
#include <cutil.h>
#include <cutil_inline.h>
#include <shrQATest.h>
#include <shrUtils.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
								

using namespace std;
enum { INPUT_, AND, NAND, OR, NOR, XOR, XNOR, NOT, FLIPFLOP};
BaseGate* all_gates;
//function declarations
__device__ bool operation(int type, bool* inputs,int numOfInputs){
	if(type == INPUT_ ){
			return inputs[0];
	}else if(type == AND){
			bool result = true;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result & inputs[i];
			}
			return result;
	}else if(type == NAND){
		bool result = true;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result & inputs[i];
			}
			return !result;
	}else if(type == OR){
			bool result = false;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result || inputs[i];
			}
			return result;
	}else if(type == NOR){
		bool result = false;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result | inputs[i];
			}
			return !result;
	}else if(type == XOR){
			bool result = false;
			for(int i = 0 ;  i < numOfInputs ; i++){
				result = result ^ inputs[i];
			}
			return result;
	}else if(type == XNOR){
			bool result = false;
			for(int i = 0 ;  i < numOfInputs ; i++){
				result = result ^ inputs[i];
			}
			return !result;
	}else if(type == NOT){
		return !inputs[0];
	}else if(type == FLIPFLOP){
		return inputs[0];
	}
	return true;
}
bool operation2(int type, bool* inputs,int numOfInputs){
	if(type == INPUT_ ){
			return inputs[0];
	}else if(type == AND){
			bool result = true;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result & inputs[i];
			}
			return result;
	}else if(type == NAND){
		bool result = true;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result & inputs[i];
			}
			return !result;
	}else if(type == OR){
			bool result = false;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result || inputs[i];
			}
			return result;
	}else if(type == NOR){
		bool result = false;
			for(int i = 0 ; i < numOfInputs ; i++){
				result = result | inputs[i];
			}
			return !result;
	}else if(type == XOR){
			bool result = false;
			for(int i = 0 ;  i < numOfInputs ; i++){
				result = result ^ inputs[i];
			}
			return result;
	}else if(type == XNOR){
			bool result = false;
			for(int i = 0 ;  i < numOfInputs ; i++){
				result = result ^ inputs[i];
			}
			return !result;
	}else if(type == NOT){
		return !inputs[0];
	}else if(type == FLIPFLOP){
		return inputs[0];
	}
	return true;
}
void define_and_set_signal(BaseGate & gate , bool signal){
	gate._defined=true;
	gate._currentOutputSignal = signal;
}
void define_and_set_nextSignal(BaseGate & gate,bool signal){
	gate._defined_nextSignal=true;
	gate._next_signalvalue = signal;
}
void  find_all_affected_gates_and_add_to_FEL(int current_time ,
																			 int max_size,
																			 bool new_value,
																			 FutureEventList &future_event_list,
																			 BaseGate * gate);
void compute_the_rest(int & counter_top,
											FutureEventList &future_event_list,
											int& current_time,
											int& max_size,
											int& time_increments);
//---------------------------------------------------------------------------------------------

//TODO for debugging only, can be deleted

void printInput(InputVector* myvector, int size){
	
	cout << "input list contains:" << endl;
	for (int i = 0 ; i < size ; i++)
	    cout << all_gates[myvector[i]._ptr_gate]._name <<" " << myvector[i]._at_time_unit << " " << myvector[i]._switches_to << endl;
}
//TODO for debugging only, can be deleted
void printCircuit(BaseGate* cGate , int index){//prints rest of the circuit starting from given gate

	int n = cGate->_currentNumberOfGates_Output;
	cout.width (index*5);
	cout << right;
	if(cGate->_defined){
		cout << "(" << cGate->_name << "-" << cGate->_currentOutputSignal  << ")" << endl;
	}else{
		cout << "(" << cGate->_name  << "-" << "undefined"  << ")" << endl;
	}
	index++;
	for(int i = 0 ; i < n ; i++){
		printCircuit(&all_gates[cGate->_outputGates[i]] ,index );
	}
}
void printCircuit( int inputSize){//prints all circuit
	for(int i = 0 ; i < inputSize ; i++){
		printCircuit(&all_gates[i], 0);
	}
}
void simulate(int current_output_gate){
				    int no_inputs=all_gates[current_output_gate]._currentNumberOfGates_Input;// get the number of inputs to XX
					bool *inputs=new bool[no_inputs];
					bool current_old_signal=all_gates[current_output_gate]._currentOutputSignal;// get the current signal of the gate XX
					for(int j=0; j<no_inputs; j++){// get all the signals to gate XX
						inputs[j]=all_gates[all_gates[current_output_gate]._inputGates[j]]._currentOutputSignal;
					}
					bool current_new_signal=operation2(all_gates[current_output_gate]._type,inputs,all_gates[current_output_gate]._numOfInputs);
					if(current_new_signal != current_old_signal){// then this gate should be added to FEL
									 all_gates[current_output_gate]._currentOutputSignal=current_new_signal;
					}
}
void simulate_DFS(int k)
{
	for(int i=0; i<all_gates[k]._currentNumberOfGates_Output ;i++){
		simulate(all_gates[k]._outputGates[i]);
		simulate_DFS(all_gates[k]._outputGates[i]);
	}
}
void simulate_all_the_circuit_with_default_values_to_inputs(int no_gates)
{
	int i=0;
	for( i=0;i<no_gates;i++){
		define_and_set_signal(all_gates[i],false);
	}
	for(i=0;i<no_gates;i++){
		simulate_DFS(i);
	}
}

BaseGate* hostGates;
BaseGate* deviceGates;
InputVector* hostInputs;
InputVector* deviceInputs;
FutureEvent* deviceFue;
//FutureEventList* hostFue;

__global__ void deney(BaseGate* tempGate,InputVector* tempVector,int a){
	
	//__shared__ BaseGate sharedGates;
	if(a > threadIdx.x ){
		tempGate[threadIdx.x]._index = a -  threadIdx.x;
	}
	tempVector[1]._at_time_unit = 7;
}
__global__ void eds(int numberOfGates,int numberOfInputGates, int maxDelay, int timeIncrement, int numberOfInputs, BaseGate* gates, InputVector* inputs){
	
	__shared__ FutureEventList *kernelFue;
	__shared__ bool **currentValues ;
	__shared__ bool change;
	__shared__ FutureEvent* tempVector;
	__shared__ int currentTime;
	__shared__ int* semaphores;
	if(threadIdx.x == numberOfGates){
		kernelFue = (FutureEventList*)malloc(sizeof(FutureEventList) * numberOfGates );
		currentValues = (bool**)malloc(sizeof(bool*) * numberOfGates);
		semaphores = (int*)malloc(sizeof(int) * numberOfGates);
		for(int i = 0; i < numberOfGates; i++){
			currentValues[i] = (bool*) malloc(sizeof(bool) * (maxDelay+1)); 
		}
		tempVector = (FutureEvent*)malloc(sizeof(FutureEvent)* numberOfInputs); 
		/* TODO make shared
		size = numberOfGates+1;
		max_Delay = maxDelay;
		time_increments = timeIncrement;*/
		currentTime = 0;
		change = true;
	}
	__syncthreads();
	if(threadIdx.x < numberOfGates){
		currentValues[threadIdx.x][currentTime%(maxDelay+1)] = gates[threadIdx.x]._currentOutputSignal;
		kernelFue[threadIdx.x].high = 0;
		kernelFue[threadIdx.x].low = 0;
		kernelFue[threadIdx.x].update = false;
		semaphores[threadIdx.x] = 1;
	}
	__syncthreads();
	if(threadIdx.x == numberOfGates){
		for(int i = 0 ; i < numberOfInputs ; i++){
			tempVector[i].time_to_change = inputs[i]._at_time_unit;
			tempVector[i].new_value = inputs[i]._switches_to;
			
			kernelFue[inputs[i]._ptr_gate].future_event_list[kernelFue[inputs[i]._ptr_gate].high] = &tempVector[i] ;
			if(++(kernelFue[inputs[i]._ptr_gate].high) == FEL_SIZE){
				kernelFue[inputs[i]._ptr_gate].high = 0;
				if(kernelFue[inputs[i]._ptr_gate].high == kernelFue[inputs[i]._ptr_gate].low){
					return;//if we enter here, then error
				}
			}
			kernelFue[inputs[i]._ptr_gate].update = true;
		}
	}
	__syncthreads();
	while(change){
		if(threadIdx.x < numberOfGates){
			if(kernelFue[threadIdx.x].update){
				if(kernelFue[threadIdx.x].future_event_list[kernelFue[threadIdx.x].low]->time_to_change ==  currentTime ){
					bool* inputs_current = (bool*)malloc(sizeof(bool) * gates[threadIdx.x]._numOfInputs);
					for(int k=0;k<gates[threadIdx.x]._numOfInputs;k++)
					{
						inputs_current[k]=currentValues[gates[threadIdx.x]._inputGates[k]][(currentTime- gates[threadIdx.x]._delay)%(maxDelay+1)];
					}
					bool newValue=operation(gates[threadIdx.x]._type,inputs_current,gates[threadIdx.x]._numOfInputs);
					gates[threadIdx.x]._currentOutputSignal=newValue;
					kernelFue[threadIdx.x].low++;
					if(kernelFue[threadIdx.x].low==FEL_SIZE){
						kernelFue[threadIdx.x].low=0;
					}
					int *outputGates=gates[threadIdx.x]._outputGates;
					int outputsize=gates[threadIdx.x]._numberOfGates_Output;
					for(int k=0; k < outputsize ; k++){
						while(atomicExch(&semaphores[outputGates[k]],0)==1);
						//critical_section
						FutureEvent *temp=(FutureEvent *)malloc(sizeof(FutureEvent));
						temp->time_to_change=currentTime+gates[outputGates[k]]._delay;
						bool check_if=false;
						for(int m=0;m<FEL_SIZE;m++){
							if(kernelFue[outputGates[k]].future_event_list[m]->time_to_change==temp->time_to_change){
								check_if=true;
								break;
							}
						}
						if(!check_if){
							kernelFue[outputGates[k]].future_event_list[kernelFue[outputGates[k]].high]=temp;
							if(++(kernelFue[outputGates[k]].high) == FEL_SIZE){
								kernelFue[outputGates[k]].high = 0;
								if(kernelFue[outputGates[k]].high== kernelFue[outputGates[k]].low){
									return;//if we enter here, then error
								}
							}
							kernelFue[outputGates[k]].update = true;
						}
						//end critical section
						atomicExch(&semaphores[outputGates[k]],1);	
					}
				}
				if(kernelFue[threadIdx.x].high == kernelFue[threadIdx.x].low){
					kernelFue[threadIdx.x].update = false;
				}			
			}
		}
		if(threadIdx.x == numberOfGates){
			currentTime++;
			for(int i = 0; i < numberOfGates; i++){
				currentValues[threadIdx.x][currentTime%(maxDelay+1)]=gates[threadIdx.x]._currentOutputSignal;
			}
			change = false;
		}
		__syncthreads();
		if(threadIdx.x < numberOfGates){
			if(kernelFue[threadIdx.x].update == true){
				change = true;	
			}
		}
	}
	
}

int main() {
	MapReader* reader = new MapReader("circuit.xml","input.xml");
	all_gates = new BaseGate [reader->getNumOfGates()];
	reader->readMap(all_gates);
	InputVector* inputs = new InputVector[reader->getNumOfInputs()];
	reader->readInput(inputs,all_gates);
	
	//FutureEventList_Constructor( *hostFue ,reader->getMaxDelay(), reader->getGcdDelay());
	
	//hipMemcpy(deviceFue,hostFue, sizeof(FutureEventList), hipMemcpyHostToDevice);
	
	hostGates = new BaseGate[reader->getNumOfGates()];
	hipMalloc((void**)&deviceGates, sizeof(BaseGate)* reader->getNumOfGates() );
	hipMemcpy(deviceGates,all_gates, sizeof(BaseGate)* reader->getNumOfGates(), hipMemcpyHostToDevice);
	
	hostInputs= new InputVector[reader->getNumOfInputs()];
	hipMalloc((void**)&deviceInputs, sizeof(InputVector)* reader->getNumOfInputs() );
	hipMemcpy(deviceInputs,inputs, sizeof(InputVector)* reader->getNumOfInputs(), hipMemcpyHostToDevice);
	
	//deney<<<1,512>>>(deviceGates,deviceInputs,reader->getNumOfGates());
	//eds<<<1,reader->getNumOfGates()>>>(reader->getNumOfGates(),reader->getNumOfInputGates(),reader->getMaxDelay(), reader->getGcdDelay(),reader->getNumOfInputs(),deviceGates,deviceInputs);
	
    
	//hipMemcpy(hostGates,deviceGates, sizeof(BaseGate)* reader->getNumOfGates(), hipMemcpyDeviceToHost);
	//hipMemcpy(hostInputs,deviceInputs, sizeof(InputVector)* reader->getNumOfInputs(), hipMemcpyDeviceToHost);
	
	//for(int i = 0; i < reader->getNumOfGates(); i++){
	//	printf("%d___%d\n", i,hostGates[i]._index);
	//}
	//printf("%d___%d\n", all_gates[1]._delay,inputs[1]._at_time_unit);
	//printf("%d___%d\n",hostGates[1]._delay, hostInputs[1]._at_time_unit);
	
		
	printInput(inputs,reader->getNumOfInputs()); //for debugging

	//timeval start, finish;
	//double elapsedTime;
	//--------------------------- START

	//gettimeofday(&start, NULL);

	//simulate_all_the_circuit_with_default_values_to_inputs(reader->getNumOfGates());

	//TODO inputList yerine inputs g�nderilecek, ve inputs size'� g�nderilecek(reader->getNumOfInputs())

	//gettimeofday(&finish, NULL);
    //----------------------------FINISH
	/*
    elapsedTime = (finish.tv_sec - start.tv_sec) * 1000.0;      // sec to ms
    elapsedTime += (finish.tv_usec - start.tv_usec) ;   // us to ms
    cout << elapsedTime << " ms.\n";*/
    //printCircuit(all_gates,reader->getNumOfGates());
	for(;;);
	return 0;
}
